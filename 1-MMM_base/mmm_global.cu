
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>
//includes for timing
#include <time.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                                       hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

#define PRINT_TIME              1
#define VALIDATE                0
#define WIDTH                 	2048
#define MATRIX_SIZE             WIDTH*WIDTH
#define TOL                     10E14

void initializeArray1D(float *arr, int len, int seed);
struct timespec diff(struct timespec start, struct timespec end);

//MMM kernel
__global__ void kernel_mmm(int width, float* M, float* N, float* P){
  const int bx_id = blockIdx.x;
  const int by_id = blockIdx.y;

  const int tx_id = threadIdx.x;
  const int ty_id = threadIdx.y;

  const int col = bx_id*blockDim.x + tx_id;
  const int row = by_id*blockDim.y + ty_id;

  float Pvalue = 0;

  for(int k=0; k<width; k++){
    float Melement = M[row*width + k];
    float Nelement = N[k*width + col];
    Pvalue += Melement*Nelement;
  }

  P[row*width + col] = Pvalue;
}



int main(int argc, char **argv){
  //struct timespec time1, time2;
  //struct timespec time_stamp;

  // GPU Timing variables
  hipEvent_t start, stop, kernel_start, kernel_stop;
  float elapsed_gpu_kernel;
  float elapsed_gpu_total;

  // Arrays on GPU global memoryc
  float *d_M;
  float *d_N;
  float *d_P;

  // Arrays on the host memory
  float *h_M;
  float *h_N;
  float *h_P;
  float *h_P_verify;

  //sum variables
  printf("Row length of the matrix = %d\n", WIDTH);

    // Select GPU
    CUDA_SAFE_CALL(hipSetDevice(0));

  // Allocate GPU memory
  size_t allocSize = MATRIX_SIZE * sizeof(float);
  CUDA_SAFE_CALL(hipMalloc((void **)&d_M, allocSize));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_N, allocSize));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_P, allocSize));

  printf("Device allocation done");


  // Allocate arrays on host memory
  h_M        = (float *) malloc(allocSize);
  h_N        = (float *) malloc(allocSize);
  h_P        = (float *) malloc(allocSize);
  h_P_verify = (float *) malloc(allocSize);

  // Initialize the host arrays
  printf("\nInitializing the arrays ...");
  // Arrays are initialized with a known seed for reproducability
  initializeArray1D(h_M, MATRIX_SIZE, 2453);
  initializeArray1D(h_N, MATRIX_SIZE, 1773);

  printf("\t... done\n\n");

#if PRINT_TIME
  // Create the cuda events
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  // Record event on the default stream
  hipEventRecord(start, 0);
#endif

  // Transfer the arrays to the GPU memory
  CUDA_SAFE_CALL(hipMemcpy(d_M, h_M, allocSize, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_N, h_N, allocSize, hipMemcpyHostToDevice));

  //block dimensions
  dim3 dimBlock(16,16,1);
  dim3 dimGrid(128,128,1); //2048
  
#if PRINT_TIME
  hipEventRecord(kernel_start, 0);
#endif

  // Launch the kernel
  kernel_mmm<<<dimGrid, dimBlock>>>(WIDTH, d_M, d_N, d_P);

#if PRINT_TIME
  // Stop and destroy the timer
  hipEventRecord(kernel_stop,0);
  hipEventSynchronize(kernel_stop);
  hipEventElapsedTime(&elapsed_gpu_kernel, kernel_start, kernel_stop);
  printf("\nMMM kernel time: %f (msec)\n", elapsed_gpu_kernel);
  hipEventDestroy(kernel_start);
  hipEventDestroy(kernel_stop);
#endif


  // Check for errors during launch
  CUDA_SAFE_CALL(hipPeekAtLastError());

  // Transfer the results back to the host
  CUDA_SAFE_CALL(hipMemcpy(h_P, d_P, allocSize, hipMemcpyDeviceToHost));

#if PRINT_TIME
  // Stop and destroy the timer
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_gpu_total, start, stop);
  printf("\nGPU time: %f (msec)\n", elapsed_gpu_total);
  hipEventDestroy(start);
  hipEventDestroy(stop);
#endif


  /*Verification using host CPU*/
#if VALIDATE
  float t;

  for(int i=0; i<WIDTH; i++){
    for(int j=0; j<WIDTH; j++){
      t = 0;
      for(int k=0; k<WIDTH; k++){
        t += h_M[i*WIDTH + k] * h_N[k*WIDTH + j];
      }
      //h_P_verify[i*WIDTH + j] += t;
      h_P_verify[i*WIDTH + j] = t;
    }
  }

  //compare two metrices
  for(int i=0; i<WIDTH; i++){
    for(int j=0; j<WIDTH; j++){
      if(abs(h_P_verify[i*WIDTH + j] - h_P[i*WIDTH + j]) > TOL){
        float diff = abs(h_P_verify[i*WIDTH + j] - h_P[i*WIDTH + j]);
        printf("Element %d,%d exceeds tolerance by %f. CPU result = %f, GPU result = %f\n", diff, i, j, h_P_verify[i*WIDTH + j], h_P[i*WIDTH + j]);
	return 1;
      } 
    }
  }
#endif

  printf("Results of CPU and GPU calculations match with a tolerance of %f\n", TOL);

  // Free-up device and host memory
  CUDA_SAFE_CALL(hipFree(d_M));
  CUDA_SAFE_CALL(hipFree(d_N));
  CUDA_SAFE_CALL(hipFree(d_P));

  free(h_M);
  free(h_N);
  free(h_P);
  free(h_P_verify);

  return 0;
}



void initializeArray1D(float *arr, int len, int seed) {
  int i;
  float randNum;
  srand(seed);

  for (i = 0; i < len; i++) {
    randNum = (float) rand();
    arr[i] = randNum;
  }
}

struct timespec diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}
