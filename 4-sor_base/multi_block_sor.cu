
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>
//#include "cuPrintf.cu"
//#include "cuPrintf.cuh"
//includes for timing
#include <time.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                                       hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

#define PRINT_TIME              1
#define VALIDATE                0
#define ROW_LEN                 2048
#define MATRIX_SIZE             ROW_LEN*ROW_LEN
#define OMEGA                   1.0

void initializeArray1D(float *arr, int len, int seed);
struct timespec diff(struct timespec start, struct timespec end);

//SOR kernel
__global__ void kernel_sor(int arrLen, float* x, float* y){
  const int bx_id = blockIdx.x;
  const int by_id = blockIdx.y;

  const int tx_id = threadIdx.x;
  const int ty_id = threadIdx.y;

  const int col = bx_id*blockDim.x + tx_id;
  const int row = by_id*blockDim.y + ty_id;


  float change;

  if((row != 0) && (row != arrLen-1) && (col != 0) && (col != arrLen-1)){
    change = x[row*arrLen + col] - 0.25 * (x[(row-1)*arrLen + col] + x[(row+1)*arrLen + col] + x[row*arrLen + col + 1] + x[row*arrLen + col - 1]);
    y[row*arrLen + col] = x[row*arrLen + col] - (change * OMEGA);
  }
  else{
    y[row*arrLen + col] = x[row*arrLen + col];
  }

}



int main(int argc, char **argv){
  struct timespec time1, time2;
  struct timespec time_stamp;

  int arrLen = 0;

  // GPU Timing variables
  hipEvent_t start, stop, kernel_start, kernel_stop;
  float elapsed_gpu, elapsed_gpu_kernel;

  // Arrays on GPU global memoryc
  float *d_x;
  float *d_y;

  // Arrays on the host memory
  float *h_x;
  float *h_y;
  float *h_result;

  //sum variables
  float gpu_sum, cpu_sum;
  
  float change;

  if (argc > 1) {
    arrLen  = atoi(argv[1]);
  }
  else {
    arrLen = ROW_LEN;
  }

  printf("Row length of the matrix = %d\n", arrLen);

    // Select GPU
    CUDA_SAFE_CALL(hipSetDevice(0));

  // Allocate GPU memory
  size_t allocSize = arrLen * arrLen * sizeof(float);
  CUDA_SAFE_CALL(hipMalloc((void **)&d_x, allocSize));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_y, allocSize));

  printf("Device allocation done");


  // Allocate arrays on host memory
  h_x                        = (float *) malloc(allocSize);
  h_y                        = (float *) malloc(allocSize);
  h_result                   = (float *) malloc(allocSize);

  // Initialize the host arrays
  printf("\nInitializing the arrays ...");
  // Arrays are initialized with a known seed for reproducability
  initializeArray1D(h_x, (arrLen*arrLen), 2453);
  printf("\t... done\n\n");

#if PRINT_TIME
  // Create the cuda events
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  // Record event on the default stream
  hipEventRecord(start, 0);
#endif

  // Transfer the arrays to the GPU memory
  CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, allocSize, hipMemcpyHostToDevice));

  //cudaPrintfInit();

  //block dimensions
  dim3 dimBlock(16,16,1);
  //dim3 dimGrid(64,64,1); //1024
  dim3 dimGrid(128,128,1); //2048
  // Launch the kernel
  hipEventRecord(kernel_start, 0);
  for(int iter=0; iter<1000; iter++){
     kernel_sor<<<dimGrid, dimBlock>>>(arrLen, d_x, d_y);
     hipDeviceSynchronize();
     kernel_sor<<<dimGrid, dimBlock>>>(arrLen, d_y, d_x);
     hipDeviceSynchronize();
  }

  // Print kernel time
#if PRINT_TIME
  hipEventRecord(kernel_stop,0);
  hipEventSynchronize(kernel_stop);
  hipEventElapsedTime(&elapsed_gpu_kernel, kernel_start, kernel_stop);
  printf("\nSOR kernel time: %f (msec)\n", elapsed_gpu_kernel);
  hipEventDestroy(kernel_start);
  hipEventDestroy(kernel_stop);
#endif

  //cudaPrintfDisplay(stdout, true);
  //cudaPrintfEnd();

  // Check for errors during launch
  CUDA_SAFE_CALL(hipPeekAtLastError());

  // Transfer the results back to the host
  CUDA_SAFE_CALL(hipMemcpy(h_result, d_x, allocSize, hipMemcpyDeviceToHost));

#if PRINT_TIME
  // Stop and destroy the timer
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_gpu, start, stop);
  printf("\nTotal GPU time: %f (msec)\n", elapsed_gpu);
  hipEventDestroy(start);
  hipEventDestroy(stop);
#endif


  /*Calculate sum*/
  gpu_sum = 0.0;

  for(int i=0; i<arrLen; i++){
    for(int j=0; j<arrLen; j++){
      gpu_sum += h_result[i*arrLen + j];
    }
  }

  //print sum
  printf("Sum of GPU calculated array elements: %f\n", gpu_sum);


  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);

  //Calculate reference result using the CPU
  /*for(int iter=0; iter<2000; iter++){
    for(int ii=0; ii<arrLen; ii+=16){
      for(int jj=0; jj<arrLen; jj+=16){
        for(int i=ii; i<ii+16; i++){
          for(int j=jj; j<jj+16; j++){
            if((i==0) || (i==arrLen-1) || (j==0) || (j==arrLen-1)){
              continue;
            }
	    else{
              change = h_x[i*arrLen + j] - 0.25 * (h_x[(i-1)*arrLen + j] + 
                                                    h_x[(i+1)*arrLen + j] + 
                                                    h_x[i*arrLen + j - 1] + 
                                                    h_x[i*arrLen + j + 1]);
	      h_x[i*arrLen + j] -= change * OMEGA;
            }
          }
        }
      }
    }
  }*/

#if VALIDATE
  for(int iter=0; iter<1000; iter++){
    // hx -> hy
    for(int i=0; i<arrLen; i++){
      for(int j=0; j<arrLen; j++){
        if((i==0) || (i==arrLen-1) || (j==0) || (j==arrLen-1)){
          h_y[i*arrLen + j] = h_x[i*arrLen + j];
        }
        else{
          change = h_x[i*arrLen + j] - 0.25 * (h_x[(i-1)*arrLen + j] + 
                                                h_x[(i+1)*arrLen + j] + 
                                                h_x[i*arrLen + j - 1] + 
                                                h_x[i*arrLen + j + 1]);
          
          h_y[i*arrLen + j] = h_x[i*arrLen + j] - (OMEGA * change);
        }
      }
    }

    // hy -> hx
    for(int i=0; i<arrLen; i++){
      for(int j=0; j<arrLen; j++){
        if((i==0) || (i==arrLen-1) || (j==0) || (j==arrLen-1)){
          h_x[i*arrLen + j] = h_y[i*arrLen + j];
        }
        else{
          change = h_y[i*arrLen + j] - 0.25 * (h_y[(i-1)*arrLen + j] + 
                                                h_y[(i+1)*arrLen + j] + 
                                                h_y[i*arrLen + j - 1] + 
                                                h_y[i*arrLen + j + 1]);
          
          h_x[i*arrLen + j] = h_y[i*arrLen + j] - (OMEGA * change);
        }
      }
    }
  }
#endif

  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
  time_stamp = diff(time1,time2);

  printf("CPU run time: %ld(ns)\n", (long int)((double)(1000000000 * time_stamp.tv_sec + time_stamp.tv_nsec)));

  //Calculate sum
  cpu_sum = 0.0;

  for(int i=0; i<arrLen; i++){
    for(int j=0; j<arrLen; j++){
      cpu_sum += h_x[i*arrLen + j];
    }
  }

  //print sum
  printf("Sum of CPU calculated array elements: %f\n", cpu_sum);



  // Free-up device and host memory
  CUDA_SAFE_CALL(hipFree(d_x));
  CUDA_SAFE_CALL(hipFree(d_y));

  free(h_x);
  free(h_y);
  free(h_result);

  return 0;
}

void initializeArray1D(float *arr, int len, int seed) {
  int i;
  float randNum;
  srand(seed);

  for (i = 0; i < len; i++) {
    randNum = (float) rand();
    arr[i] = randNum;
  }
}

struct timespec diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}
